/*
EE 451
Course Project: Raytracer
Serial Version

Names: James Lee, Darwin Mendyke, Ahsan Zaman
*/


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cmath>
#include <iostream>
#include <fstream>
#include <vector>
#include <string.h>
#include <time.h>

using namespace std;

#define MAX_TRIANGLES 2000
#define MAX_SPHERES 10
#define MAX_LIGHTS 10

char *filename=0;

//bool done = false;

//you may want to make these smaller for debugging purposes
#define WIDTH 640
#define HEIGHT 480

//the field of view of the camera
#define fov 60.0

unsigned char buffer[HEIGHT][WIDTH][3];

struct Vertex
{
  double position[3];
  double color_diffuse[3];
  double color_specular[3];
  double normal[3];
  double shininess;
};

typedef struct _Triangle
{
  struct Vertex v[3];
} Triangle;

typedef struct _Sphere
{
  double position[3];
  double color_diffuse[3];
  double color_specular[3];
  double shininess;
  double radius;
} Sphere;

typedef struct _Light
{
  double position[3];
  double color[3];
} Light;

__device__ Triangle* triangles;//[MAX_TRIANGLES];
__device__ Sphere* spheres;//[MAX_SPHERES];
__device__ Light* lights;//[MAX_LIGHTS];
__device__ double* ambient_light;//[3];

__device__ int* num_triangles;//=0;
__device__ int* num_spheres;//=0;
__device__ int* num_lights;//=0;

// Helper function to normalize a given vecter to a certain length, typically 1
__device__
void normalize(double p[3])
{
   double distance = fmax(sqrt(pow(p[0], 2.0) + pow(p[1], 2.0) + pow(p[2], 2.0)), 0.001);
   p[0] = p[0] / distance;
   p[1] = p[1] / distance;
   p[2] = p[2] / distance;
}

// Vector arithmitec function to perform vector subtraction
__device__
void subtract(double v0[3], double v1[3], double result[3])
{
   for (int i = 0; i < 3; i++) result[i] = v0[i] - v1[i];
}

// Vector arithmitec function to perform vector multiplication
__device__
void multiply(double v[3], double s, double result[3])
{
   for (int i = 0; i < 3; i++) result[i] = v[i] * s;
}

// Vector arithmitec function to calculate the vector cross product
__device__
void cross(double v0[3], double v1[3], double result[3])
{
   result[0] = (v0[1] * v1[2]) - (v1[1] * v0[2]);
   result[1] = (v1[0] * v0[2]) - (v0[0] * v1[2]);
   result[2] = (v0[0] * v1[1]) - (v1[0] * v0[1]);
}

// Vector arithmitec function to calculate the vector dot product
__device__
double dot(double v0[3], double v1[3])
{
   return (v0[0] * v1[0]) + (v0[1] * v1[1]) + (v0[2] * v1[2]);
}

// Helper function to compare distances between two points relative to the origin.
// Sets inter with the closer point
__device__
bool compDistances(double o[3], double (&inter)[3], double (&newInter)[3])
{
   double a[3] = {inter[0] - o[0], inter[1] - o[1], inter[2] - o[2]};
   double b[3] = {newInter[0] - o[0], newInter[1] - o[1], newInter[2] - o[2]};
   if (fmax(sqrt(pow(b[0], 2.0) + pow(b[1], 2.0) + pow(b[2], 2.0)), 0.001) < fmax(sqrt(pow(a[0], 2.0) + pow(a[1], 2.0) + pow(a[2], 2.0)), 0.001))
   {
      for (int i = 0; i < 3; i++) inter[i] = newInter[i];
      return true;
   }
   return false;
}

// Given a sphere and a ray, determines if there is an intersection.
// If so, stores the coords of the intersection
__device__
bool intersectsSphere(Sphere sphere, double o[3], double d[3], double (&intersection)[3])
{
   double a = 1.0;
   double b = 2 * (d[0] * (o[0] - sphere.position[0]) + d[1] * (o[1] - sphere.position[1]) + d[2] * (o[2] - sphere.position[2]));
   double c = pow((o[0] - sphere.position[0]), 2.0) + pow((o[1] - sphere.position[1]), 2.0) + pow((o[2] - sphere.position[2]), 2.0) - pow(sphere.radius, 2.0);
   double t0 = (-b + sqrt(pow(b, 2.0) - (4.0 * a * c))) / 2.0;
   double t1 = (-b - sqrt(pow(b, 2.0) - (4.0 * a * c))) / 2.0;
   if (fmin(t0, t1) > 0)
   {
      double newIntersection[3] = {o[0] + fmin(t0, t1) * d[0], o[1] + fmin(t0, t1) * d[1], o[2] + fmin(t0, t1) * d[2]};

      if (compDistances(o, intersection, newIntersection)) return true;
   }
   return false;
}

// Given a triangle shape and a ray, determines if there is an intersection, using examples from the above link
__device__
bool intersectsTriangle(Triangle triangle, double o[3], double d[3], double (&intersection)[3], double (&bcoords)[3])
{
   double v0v1[3], v0v2[3], pvec[3], tvec[3], qvec[3], t, u, v;
   subtract(triangle.v[1].position, triangle.v[0].position, v0v1);
   subtract(triangle.v[2].position, triangle.v[0].position, v0v2);

   cross(d, v0v2, pvec);
   double det = dot(v0v1, pvec);

   double invDet = 1 / det;

   subtract(o, triangle.v[0].position, tvec);
   u = dot(tvec, pvec) * invDet;
   if (u < 0 || u > 1) return false;

   cross(tvec, v0v1, qvec);
   v = dot(d, qvec) * invDet;
   if (v < 0 || u + v > 1) return false;

   t = dot(v0v2, qvec) * invDet;
   if (t <= 0) return false; // checks if intersection is in positive ray direction or negative

   double newIntersection[3] = {o[0] + t * d[0], o[1] + t * d[1], o[2] + t * d[2]};
   if (compDistances(o, intersection, newIntersection))
   {
      bcoords[0] = u;
      bcoords[1] = v;
      bcoords[2] = 1.0 - bcoords[0] - bcoords[1];
      return true;
   }

   return false;
}


// Recursive function to perform ray tracing given a ray.
__device__
void trace(double o[3], double d[3], int num, double* trace_result){
   bool intersectTriangle = false, intersectSphere = false;
   double bcoords[3], other1[3], intersection[3] = {1000.0, 1000.0, 1000.0};
   int index = 0;

   // looks for an intersection between the input ray and a shape
   for (int i = 0; i < *num_triangles; i++) if (intersectsTriangle(triangles[i], o, d, intersection, bcoords)) intersectTriangle = true, index = i;
   for (int i = 0; i < *num_spheres; i++) if (intersectsSphere(spheres[i], o, d, intersection)) intersectSphere = true, index = i;

   // returns background color if no intersections or if reach max recursive call
   if ((!intersectTriangle && !intersectSphere) || num > 2){
    trace_result[0] = 1.0;
    trace_result[1] = 1.0;
    trace_result[2] = 1.0;
   }

   double illumination[3] = {ambient_light[0], ambient_light[1], ambient_light[2]};
   double l[3], n[3], n1[3], v[3], r[3], recursive_r[3], diffuse[3], specular[3], shiny;

   // iterates through each light in the scene
   for (int j = 0; j < *num_lights; j++)
   {
      for (int i = 0; i < 3; i++)
      {
         l[i] = lights[j].position[i] - intersection[i];
         v[i] = o[i] - intersection[i];
      }
      if (intersectSphere)
      {
         for (int i = 0; i < 3; i++)
         {
            n[i] = intersection[i] - spheres[index].position[i];
            diffuse[i] = spheres[index].color_diffuse[i];
            specular[i] = spheres[index].color_specular[i];
         }
         shiny = spheres[index].shininess;
      }
      else if (intersectTriangle)
      {
         Triangle shape = triangles[index];
         for (int i = 0; i < 3; i++)
         {
            n[i] = shape.v[0].normal[i] * bcoords[2] + shape.v[1].normal[i] * bcoords[0] + shape.v[2].normal[i] * bcoords[1];
            diffuse[i] = shape.v[0].color_diffuse[i] * bcoords[2] + shape.v[1].color_diffuse[i] * bcoords[0] + shape.v[2].color_diffuse[i] * bcoords[1];
            specular[i] = shape.v[0].color_specular[i] * bcoords[2] + shape.v[1].color_specular[i] * bcoords[0] + shape.v[2].color_specular[i] * bcoords[1];
         }
         shiny = shape.v[0].shininess * bcoords[2] + shape.v[1].shininess * bcoords[0] + shape.v[2].shininess * bcoords[1];
      }

      // initializes the shadow ray from the intersection point
      double normalized_pos[3] = {lights[j].position[0] - intersection[0], lights[j].position[1] - intersection[1], lights[j].position[2] - intersection[2]};
      double shadowIntersection[3] = {lights[j].position[0], lights[j].position[1], lights[j].position[2]};
      double shadowOrigin[3] = {intersection[0] + 0.001 * n[0], intersection[1] + 0.001 * n[1], intersection[2] + 0.001 * n[2]};
      bool shadow = false;
      normalize(normalized_pos);

      // checks if the shadow ray intersects with a shape
      for (int i = 0; i < *num_spheres; i++)
         if (intersectsSphere(spheres[i], shadowOrigin, normalized_pos, shadowIntersection)) shadow = true;
      for (int i = 0; i < *num_triangles; i++)
         if (intersectsTriangle(triangles[i], shadowOrigin, normalized_pos, shadowIntersection, other1)) shadow = true;

      normalize(l);
      normalize(n);
      normalize(v);
      multiply(n, 2 * dot(l, n), n1);
      subtract(n1, l, r);
      normalize(r); // calculates the reflection ray

      // if there is no shadow at the point, calculates illumination using phong shading equation
      if (!shadow)
      {
         for (int i = 0; i < 3; i++)
         {
            double a = diffuse[i] * fmax(0.0, dot(l, n));
            double b = specular[i] * pow(fmax(0.0, dot(v, r)), shiny);
            illumination[i] += lights[j].color[i] * (a + b);
            illumination[i] = fmin(illumination[i], 1.0);
         }
      }
   }
   // return illumination;
   // uncomment this code below to recursively call tracer function on reflection ray
   multiply(n, 2 * dot(v, n), n1);
   subtract(n1, v, recursive_r);
   double recursiveOrigin[3] = {intersection[0] + 0.01 * recursive_r[0], intersection[1] + 0.01 * recursive_r[1], intersection[2] + 0.01 * recursive_r[2]};
   normalize(recursive_r);
   trace(recursiveOrigin, recursive_r, ++num, trace_result);
   for (int i = 0; i < 3; i++)
   {
      trace_result[i] = (1 - specular[i]) * illumination[i] + specular[i] * trace_result[i];
   }
}

// Iterates through each pixel on the window and generates a ray, which it passes to the tracer function
__global__
void draw_scene(double* result)
{
  double focalLength = 0.5 * WIDTH * sqrt(3.0) * 0.75;
  double origin[3] = {0, 0, 0};
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  double direction[3] = {x - ((double) WIDTH / 2.0), y - ((double) HEIGHT / 2.0), -1 * focalLength};
  normalize(direction);
  double* color = new double[3];
  trace(origin, direction, 0, color);
  result[1]=255*1;//color[0];
  result[2]=255*1;//color[1];
  result[3]=255*1;//color[2];
}

void parse_check(char *expected,char *found)
{
  if(strcasecmp(expected,found))
    {
      // char error[100];
      printf("Expected '%s ' found '%s '\n",expected,found);
      printf("Parse error, abnormal abortion\n");
      exit(0);
    }

}

void parse_doubles(FILE*file, char *check, double p[3])
{
  char str[100];
  fscanf(file,"%s",str);
  parse_check(check,str);
  fscanf(file,"%lf %lf %lf",&p[0],&p[1],&p[2]);
  // printf("%s %lf %lf %lf\n",check,p[0],p[1],p[2]);
}

void parse_rad(FILE*file,double *r)
{
  char str[100];
  fscanf(file,"%s",str);
  parse_check((char *)"rad:",str);
  fscanf(file,"%lf",r);
  // printf("rad: %f\n",*r);
}

void parse_shi(FILE*file,double *shi)
{
  char s[100];
  fscanf(file,"%s",s);
  parse_check((char *)"shi:",s);
  fscanf(file,"%lf",shi);
  // printf("shi: %f\n",*shi);
}

int loadScene(char *argv, int num_triangles_, int num_spheres_, int num_lights_, Triangle* triangles_, Sphere* spheres_, Light* lights_, double* ambient_light_)
{
  FILE *file = fopen(argv,"r");
  int number_of_objects;
  char type[50];
  int i;
  Triangle t;
  Sphere s;
  Light l;
  fscanf(file,"%i",&number_of_objects);

  // printf("number of objects: %i\n",number_of_objects);
  // char str[200];

  parse_doubles(file,(char *)"amb:",ambient_light_);

  for(i=0;i < number_of_objects;i++)
  {
    fscanf(file,"%s\n",type);
    // printf("%s\n",type);
    if(strcasecmp(type,"triangle")==0)
    {
      // printf("found triangle\n");
      int j;
      for(j=0;j < 3;j++)
      {
        parse_doubles(file,(char *)"pos:",t.v[j].position);
        parse_doubles(file,(char *)"nor:",t.v[j].normal);
        parse_doubles(file,(char *)"dif:",t.v[j].color_diffuse);
        parse_doubles(file,(char *)"spe:",t.v[j].color_specular);
        parse_shi(file,&t.v[j].shininess);
      }
      if(num_triangles_ == MAX_TRIANGLES)
      {
        printf("too many triangles, you should increase MAX_TRIANGLES!\n");
        exit(0);
      }
      triangles_[num_triangles_++] = t;
    }
    else if(strcasecmp(type,"sphere")==0)
    {
      // printf("found sphere\n");
      parse_doubles(file,(char *)"pos:",s.position);
      parse_rad(file,&s.radius);
      parse_doubles(file,(char *)"dif:",s.color_diffuse);
      parse_doubles(file,(char *)"spe:",s.color_specular);
      parse_shi(file,&s.shininess);
      if(num_spheres_ == MAX_SPHERES) {
        printf("too many spheres, you should increase MAX_SPHERES!\n");
        exit(0);
      }
      spheres_[num_spheres_++] = s;
    }
    else if(strcasecmp(type,"light")==0)
    {
      // printf("found light\n");
      parse_doubles(file,(char *)"pos:",l.position);
      parse_doubles(file,(char *)"col:",l.color);
      if(num_lights_ == MAX_LIGHTS){
        printf("too many lights, you should increase MAX_LIGHTS!\n");
        exit(0);
      }
      lights_[num_lights_++] = l;
    }
    else{
      printf("unknown type in scene description:\n%s\n",type);
      exit(0);
    }
  }
  fclose(file);
  return 0;
}
void make_bitmap(double* rgbVals, char* fileToWrite)
{
  typedef struct                       /**** BMP file header structure ****/
      {
      unsigned int   bfSize;           /* Size of file */
      unsigned short bfReserved1;      /* Reserved */
      unsigned short bfReserved2;      /* ... */
      unsigned int   bfOffBits;        /* Offset to bitmap data */
      } BITMAPFILEHEADER;

  typedef struct                       /**** BMP file info structure ****/
      {
      unsigned int   biSize;           /* Size of info header */
      int            biWidth;          /* Width of image */
      int            biHeight;         /* Height of image */
      unsigned short biPlanes;         /* Number of color planes */
      unsigned short biBitCount;       /* Number of bits per pixel */
      unsigned int   biCompression;    /* Type of compression to use */
      unsigned int   biSizeImage;      /* Size of image data */
      int            biXPelsPerMeter;  /* X pixels per meter */
      int            biYPelsPerMeter;  /* Y pixels per meter */
      unsigned int   biClrUsed;        /* Number of colors used */
      unsigned int   biClrImportant;   /* Number of important colors */
      } BITMAPINFOHEADER;

  BITMAPFILEHEADER bfh;
  BITMAPINFOHEADER bih;

  /* Magic number for file. It does not fit in the header structure due to alignment requirements, so put it outside */
  unsigned short bfType=0x4d42;
  bfh.bfReserved1 = 0;
  bfh.bfReserved2 = 0;
  bfh.bfSize = 2+sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER)+640*480*3;
  bfh.bfOffBits = 0x36;

  bih.biSize = sizeof(BITMAPINFOHEADER);
  bih.biWidth = WIDTH;
  bih.biHeight = HEIGHT;
  bih.biPlanes = 1;
  bih.biBitCount = 24;
  bih.biCompression = 0;
  bih.biSizeImage = 0;
  bih.biXPelsPerMeter = 5000;
  bih.biYPelsPerMeter = 5000;
  bih.biClrUsed = 0;
  bih.biClrImportant = 0;

  FILE *file = fopen(fileToWrite, "wb");
  if (!file)
      {
      printf("Could not write file\n");
      return;
      }

  /*Write headers*/
  fwrite(&bfType,1,sizeof(bfType),file);
  fwrite(&bfh, 1, sizeof(bfh), file);
  fwrite(&bih, 1, sizeof(bih), file);

  /*Write bitmap*/
  for (int y=0; y<bih.biHeight; y++)
      {
      for (int x = 0; x < bih.biWidth; x++)
          {
          /*compute some pixel values*/
          unsigned char r = rgbVals[x*y*1];
          unsigned char g = rgbVals[x*y*2];
          unsigned char b = rgbVals[x*y*3];
          fwrite(&b, 1, 1, file);
          fwrite(&g, 1, 1, file);
          fwrite(&r, 1, 1, file);
          }
      }
  fclose(file);   
}

inline bool exists_file(char* name){
  if(FILE *file = fopen(name, "r")){
      fclose(file);
      return true;
  }
  else return false;
}

int main (int argc, char ** argv)
{
  if (argc<3 || argc > 3){
    printf ("usage: %s <scenefile> <bmp_name>\n", argv[0]);
    exit(0);
  }
  char* fileToRead = argv[1];
  char* fileToWrite = argv[2];

  if(!exists_file(fileToRead)){
    cout << "Input file does not exist.\n" << endl;
    exit(0);
  }
  int num_triangles_=0;
  int num_spheres_=0;
  int num_lights_=0;
  Triangle triangles_[MAX_TRIANGLES];
  Sphere spheres_[MAX_SPHERES];
  Light lights_[MAX_LIGHTS];
  double ambient_light_[3];
  loadScene(fileToRead, num_triangles_, num_spheres_, num_lights_, triangles_, spheres_, lights_, ambient_light_);

  double* drawing;
  double time;
  struct timespec start, stop;
  int GRID_DIM = WIDTH;
  int BLOCK_DIM = HEIGHT;
  if( clock_gettime(CLOCK_REALTIME, &start) == -1) { perror("clock gettime");}
  
  //copy stuff over
  hipMallocManaged(&drawing, WIDTH*HEIGHT*3*sizeof(double));

  hipMemcpy(triangles, triangles_, MAX_TRIANGLES*sizeof( Triangle), hipMemcpyHostToDevice);
  hipMemcpy(spheres, spheres_, MAX_SPHERES*sizeof( Sphere), hipMemcpyHostToDevice);
  hipMemcpy(lights, lights_, MAX_LIGHTS*sizeof( Light), hipMemcpyHostToDevice);
  hipMemcpy(ambient_light, ambient_light_, 3*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(num_triangles, &num_triangles_, 1*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(num_spheres, &num_spheres_, 1*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(num_lights, &num_lights_, 1*sizeof(int), hipMemcpyHostToDevice);

  // local_drawing = new double[WIDTH*HEIGHT*3];
  //measure how long it takes to render the image
  // cudaMemcpy(drawing, local_drawing, WIDTH*HEIGHT*sizeof(double), cudaMemcpyDeviceToHost);
  draw_scene<<<GRID_DIM, BLOCK_DIM>>>(drawing);
  hipDeviceSynchronize();
  cout << drawing[1] << drawing[2] << drawing[2] << endl;
  // cudaMemcpy(drawing, local_drawing, WIDTH*HEIGHT*sizeof(double), cudaMemcpyDeviceToHost);
  if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror("clock gettime");}
  time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
  printf("Execution time for %s: %f seconds.\n",fileToRead, time);

  make_bitmap(drawing, fileToWrite);
  
  hipFree(drawing);
  hipFree(triangles);
  hipFree(spheres);
  hipFree(lights);
  hipFree(ambient_light);
  hipFree(num_triangles);
  hipFree(num_spheres);
  hipFree(num_lights);
}
